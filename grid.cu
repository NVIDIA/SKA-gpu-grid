#include "hip/hip_runtime.h"
#include <iostream>
#include "math.h"
#include "stdlib.h"

#include "grid_gpu.cuh"
#include "Defines.h"
#include "hip/hip_runtime.h"

//With managed memory, grid.cpp must be compiled as CUDA
//in which case float2 and double2 are predefined.
//typedef struct {float x,y;} float2;
//typedef struct {double x,y;} double2;

#define single 77
#if PRECISION==single
#define PRECISION float
#endif

#ifndef PRECISION
#define PRECISION double
#endif
#define PASTER(x) x ## 2
#define EVALUATOR(x) PASTER(x)
#define PRECISION2 EVALUATOR(PRECISION)


void init_gcf(PRECISION2 *gcf, size_t size) {

  for (size_t sub_x=0; sub_x<GCF_GRID; sub_x++ )
   for (size_t sub_y=0; sub_y<GCF_GRID; sub_y++ )
    for(size_t x=0; x<size; x++)
     for(size_t y=0; y<size; y++) {
       //Some nonsense GCF
       PRECISION tmp = sin(6.28*x/size/GCF_GRID)*exp(-(1.0*x*x+1.0*y*y*sub_y)/size/size/2);
       gcf[size*size*(sub_x+sub_y*GCF_GRID)+x+y*size].x = tmp*sin(1.0*x*sub_x/(y+1));
       gcf[size*size*(sub_x+sub_y*GCF_GRID)+x+y*size].y = tmp*cos(1.0*x*sub_x/(y+1));
       //std::cout << tmp << gcf[x+y*size].x << gcf[x+y*size].y << std::endl;
     }

}

void gridCPU(PRECISION2* out, PRECISION2 *in, size_t npts, PRECISION2 *img, size_t img_dim, PRECISION2 *gcf, size_t gcf_dim) {
//grid on the CPU
//  out (out) - the output values for each location
//  in  (in)  - the locations to be interpolated 
//  npts (in) - number of locations
//  img (in) - the image
//  img_dim (in) - dimension of the image
//  gcf (in) - the gridding convolution function
//  gcf_dim (in) - dimension of the GCF

   //offset gcf to point to the middle for cleaner code later
   gcf += GCF_DIM*(GCF_DIM+1)/2;
#pragma acc parallel loop copyout(out[0:NPOINTS]) copyin(in[0:NPOINTS],gcf[0:GCF_GRID*GCF_GRID*GCF_DIM*GCF_DIM],img[IMG_SIZE*IMG_SIZE]) gang
#pragma omp parallel for
   for(size_t n=0; n<NPOINTS; n++) {
      //std::cout << "in = " << in[n].x << ", " << in[n].y << std::endl;
      int sub_x = floorf(GCF_GRID*(in[n].x-floorf(in[n].x)));
      int sub_y = floorf(GCF_GRID*(in[n].y-floorf(in[n].y)));
      //std::cout << "sub = "  << sub_x << ", " << sub_y << std::endl;
      int main_x = floor(in[n].x); 
      int main_y = floor(in[n].y); 
      //std::cout << "main = " << main_x << ", " << main_y << std::endl;
      PRECISION sum_r = 0.0;
      PRECISION sum_i = 0.0;
      #pragma acc parallel loop collapse(2) reduction(+:sum_r,sum_i) vector
#pragma omp parallel for collapse(2) reduction(+:sum_r, sum_i)
      for (int a=-GCF_DIM/2; a<GCF_DIM/2 ;a++)
      for (int b=-GCF_DIM/2; b<GCF_DIM/2 ;b++) {
         PRECISION r1 = img[main_x+a+IMG_SIZE*(main_y+b)].x; 
         PRECISION i1 = img[main_x+a+IMG_SIZE*(main_y+b)].y; 
         PRECISION r2 = gcf[GCF_DIM*GCF_DIM*(GCF_GRID*sub_y+sub_x) + 
                        GCF_DIM*b+a].x;
         PRECISION i2 = gcf[GCF_DIM*GCF_DIM*(GCF_GRID*sub_y+sub_x) + 
                        GCF_DIM*b+a].y;
         if (main_x+a < 0 || main_y+b < 0 || 
             main_x+a >= IMG_SIZE  || main_y+b >= IMG_SIZE) {
         } else {
            sum_r += r1*r2 - i1*i2; 
            sum_i += r1*i2 + r2*i1;
         }
      }
      out[n].x = sum_r;
      out[n].y = sum_i;
      //std::cout << "val = " << out[n].r << "+ i" << out[n].i << std::endl;
   } 
   gcf -= GCF_DIM*(GCF_DIM+1)/2;
}
template <class T,class Thalf>
int w_comp_main(const void* A, const void* B) {
   Thalf quota, rema, quotb, remb;
   rema = modf((*((T*)A)).x, &quota);
   remb = modf((*((T*)B)).x, &quotb);
   if (quota > quotb) return 1;
   if (quota < quotb) return -1;
   else {
     rema = modf((*((T*)A)).y, &quota);
     remb = modf((*((T*)B)).y, &quotb);
     if (quota > quotb) return 1;
     if (quota < quotb) return -1;
     else return 0;
   }
   return 0;
}
template <class T,class Thalf>
int w_comp_sub(const void* A, const void* B) {
   Thalf quota, rema, quotb, remb;
   rema = modf((*((T*)A)).x, &quota);
   remb = modf((*((T*)B)).x, &quotb);
   int sub_xa = (int) (GCF_GRID*rema);
   int sub_xb = (int) (GCF_GRID*remb);
   rema = modf((*((T*)A)).y, &quota);
   remb = modf((*((T*)B)).y, &quotb);
   int suba = (int) (GCF_GRID*rema) + GCF_GRID*sub_xa;
   int subb = (int) (GCF_GRID*remb) + GCF_GRID*sub_xb;
   if (suba > subb) return 1;
   if (suba < subb) return -1;
   return 0;
}
template <class T,class Thalf>
int w_comp_full(const void* A, const void* B) {
   int result = w_comp_sub<T,Thalf>(A,B);
   if (0==result) return w_comp_main<T,Thalf>(A,B);
   else return result;
}
#if 0
struct comp_grid {
   int blockgrid, blocksize;
   public:
   comp_grid(int img_dim, int gcf_dim) {
      blocksize = gcf_dim/2;
      blockgrid = img_dim/blocksize;
   }
   int __cdecl operator () (const void* A, const void* B) const {
      int gridxa = (*(int2*)A).x/GCF_GRID;
      int gridxb = (*(int2*)B).x/GCF_GRID;
      int gridya = (*(int2*)A).y/GCF_GRID;
      int gridyb = (*(int2*)B).y/GCF_GRID;
      if (gridya > gridyb) return 1;
      if (gridya < gridyb) return -1;
      if (gridxa > gridxb) return 1;
      if (gridxa < gridxb) return  -1;
      int suba = GCF_GRID*((*(int2*)A).x%GCF_GRID) + (*(int2*)A).y%GCF_GRID;
      int subb = GCF_GRID*((*(int2*)B).x%GCF_GRID) + (*(int2*)B).y%GCF_GRID;
      if (suba > subb) return 1;
      if (suba < subb) return -1;
      return  0;
   }
};
#else
template <class T, class Thalf>
int comp_grid (const void* A, const void* B) {
      int blocksize = GCF_DIM/2;
      int mainxa = floorf((*(T*)A).x);
      int mainxb = floorf((*(T*)B).x);
      int mainya = floorf((*(T*)A).y);
      int mainyb = floorf((*(T*)B).y);
      int gridxa = mainxa/blocksize;
      int gridxb = mainxb/blocksize;
      int gridya = mainya/blocksize;
      int gridyb = mainyb/blocksize;
      if (gridya*IMG_SIZE/blocksize+gridxa > 
          gridyb*IMG_SIZE/blocksize+gridxb) return 1;
      if (gridya*IMG_SIZE/blocksize+gridxa < 
          gridyb*IMG_SIZE/blocksize+gridxb) return -1;
      Thalf suba = GCF_GRID*((*(T*)A).x-mainxa) + (*(T*)A).y-mainya;
      Thalf subb = GCF_GRID*((*(T*)B).x-mainxb) + (*(T*)B).y-mainyb;
      if (suba > subb) return 1;
      if (suba < subb) return -1;
      return  0;
}
#endif


int main(void) {

#ifdef __MANAGED
   PRECISION2* out, *in, *img, *gcf;
   hipMallocManaged(&out, sizeof(PRECISION2)*NPOINTS);
   hipMallocManaged(&in, sizeof(PRECISION2)*NPOINTS);
   hipMallocManaged(&img, sizeof(PRECISION2)*(IMG_SIZE*IMG_SIZE+2*IMG_SIZE*GCF_DIM+2*GCF_DIM));
   hipMallocManaged(&gcf, sizeof(PRECISION2)*64*GCF_DIM*GCF_DIM);
#else
   PRECISION2* out = (PRECISION2*) malloc(sizeof(PRECISION2)*NPOINTS);
   PRECISION2* in = (PRECISION2*) malloc(sizeof(PRECISION2)*NPOINTS);
   PRECISION2 *img = (PRECISION2*) malloc((IMG_SIZE*IMG_SIZE+2*IMG_SIZE*GCF_DIM+2*GCF_DIM)*sizeof(PRECISION2));

   PRECISION2 *gcf = (PRECISION2*) malloc(64*GCF_DIM*GCF_DIM*sizeof(PRECISION2));
#endif

   //img is padded (above and below) to avoid overruns
   img += IMG_SIZE*GCF_DIM+GCF_DIM;
    
   init_gcf(gcf, GCF_DIM);
   srand(2541617);
   for(size_t n=0; n<NPOINTS; n++) {
      in[n].x = ((float)rand())/RAND_MAX*8000;
      in[n].y = ((float)rand())/RAND_MAX*8000;
   }
   for(size_t x=0; x<IMG_SIZE;x++)
   for(size_t y=0; y<IMG_SIZE;y++) {
      img[x+IMG_SIZE*y].x = exp(-((x-1400.0)*(x-1400.0)+(y-3800.0)*(y-3800.0))/8000000.0)+1.0;
      img[x+IMG_SIZE*y].y = 0.4;
   }
   //Zero the data in the offset areas
   for (int x=-IMG_SIZE*GCF_DIM-GCF_DIM;x<0;x++) {
      img[x].x = 0.0; img[x].y = 0.0;
   }
   for (int x=0;x<IMG_SIZE*GCF_DIM+GCF_DIM;x++) {
      img[x+IMG_SIZE*IMG_SIZE].x = 0.0; img[x+IMG_SIZE*IMG_SIZE].y = 0.0;
   }

#ifdef __SCATTER
   std::qsort(in, NPOINTS, sizeof(PRECISION2), comp_grid<PRECISION2,PRECISION>);
#else
#ifdef __MOVING_WINDOW
   std::qsort(in, NPOINTS, sizeof(PRECISION2), w_comp_main<PRECISION2,PRECISION>);
#else
   std::qsort(in, NPOINTS, sizeof(PRECISION2), w_comp_sub<PRECISION2,PRECISION>);
#endif
#endif
   std::cout << "sorted" << std::endl;
   
   gridGPU(out,in,NPOINTS,img,IMG_SIZE,gcf,GCF_DIM);
#ifdef __CPU_CHECK
   std::cout << "Computing on CPU..." << std::endl;
   PRECISION2 *out_cpu=(PRECISION2*)malloc(sizeof(PRECISION2)*NPOINTS);
   gridCPU(out_cpu,in,NPOINTS,img,IMG_SIZE,gcf,GCF_DIM);
#endif


#ifdef __CPU_CHECK
   std::cout << "Checking results against CPU:" << std::endl;
   for (size_t n = 0; n < NPOINTS; n++) {
     if (fabs(out[n].x-out_cpu[n].x) > 0.0000001 ||
         fabs(out[n].y-out_cpu[n].y) > 0.0000001 )
        std::cout << n << ": F(" << in[n].x << ", " << in[n].y << ") = " 
                  << out[n].x << ", " << out[n].y 
                  << " vs. " << out_cpu[n].x << ", " << out_cpu[n].y 
                  << std::endl;
   }
#endif
   img -= GCF_DIM + IMG_SIZE*GCF_DIM;
#ifdef __MANAGED
   hipFree(out);
   hipFree(in);
   hipFree(img);
   hipFree(gcf);
#else
   free(out);
   free(in);
   free(img);
   free(gcf);
#endif
}
