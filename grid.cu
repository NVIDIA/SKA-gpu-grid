#include "hip/hip_runtime.h"
#include <iostream>
#include "math.h"
#include "stdlib.h"

#include "grid_gpu.cuh"
#include "Defines.h"
#include "hip/hip_runtime.h"

//With managed memory, grid.cpp must be compiled as CUDA
//in which case float2 and double2 are predefined.
//typedef struct {float x,y;} float2;
//typedef struct {double x,y;} double2;

#define single 77
#if PRECISION==single
#define PRECISION float
#endif

#ifndef PRECISION
#define PRECISION double
#endif
#define PASTER(x) x ## 2
#define EVALUATOR(x) PASTER(x)
#define PRECISION2 EVALUATOR(PRECISION)


void init_gcf(PRECISION2 *gcf, size_t size) {

  for (size_t sub_x=0; sub_x<GCF_GRID; sub_x++ )
   for (size_t sub_y=0; sub_y<GCF_GRID; sub_y++ )
    for(size_t x=0; x<size; x++)
     for(size_t y=0; y<size; y++) {
       //Some nonsense GCF
       PRECISION tmp = sin(6.28*x/size/GCF_GRID)*exp(-(1.0*x*x+1.0*y*y*sub_y)/size/size/2);
       gcf[size*size*(sub_x+sub_y*GCF_GRID)+x+y*size].x = tmp*sin(1.0*x*sub_x/(y+1))+0.4;
       gcf[size*size*(sub_x+sub_y*GCF_GRID)+x+y*size].y = tmp*cos(1.0*x*sub_x/(y+1))-0.2;
       //std::cout << tmp << gcf[x+y*size].x << gcf[x+y*size].y << std::endl;
     }

}

void gridCPU(PRECISION2* out, PRECISION2 *in, PRECISION2 *in_vals, size_t npts, size_t img_dim, PRECISION2 *gcf, size_t gcf_dim) {
//degrid on the CPU
//  out (out) - the output image
//  in  (in)  - the input locations
//  in_vals (in) - input values
//  npts (in) - number of locations
//  img_dim (in) - dimension of the image
//  gcf (in) - the gridding convolution function
//  gcf_dim (in) - dimension of the GCF

   //Zero the output
   for (size_t n=0;n<IMG_SIZE*IMG_SIZE; n++) out[n].x = out[n].y = 0.0;
   //offset gcf to point to the middle for cleaner code later
   gcf += GCF_DIM*(GCF_DIM+1)/2;
//#pragma acc parallel loop copyout(out[0:NPOINTS]) copyin(in[0:NPOINTS],gcf[0:GCF_GRID*GCF_GRID*GCF_DIM*GCF_DIM],img[IMG_SIZE*IMG_SIZE]) gang
//#pragma omp parallel for
   for(size_t n=0; n<NPOINTS; n++) {
      //std::cout << "in = " << in[n].x << ", " << in[n].y << std::endl;
      int sub_x = floorf(GCF_GRID*(in[n].x-floorf(in[n].x)));
      int sub_y = floorf(GCF_GRID*(in[n].y-floorf(in[n].y)));
      //std::cout << "sub = "  << sub_x << ", " << sub_y << std::endl;
      int main_x = floor(in[n].x); 
      int main_y = floor(in[n].y); 
      //std::cout << "main = " << main_x << ", " << main_y << std::endl;
//      #pragma acc parallel loop collapse(2) reduction(+:sum_r,sum_i) vector
//#pragma omp parallel for collapse(2) reduction(+:sum_r, sum_i)
      for (int a=GCF_DIM/2; a>=-GCF_DIM/2 ;a--)
      for (int b=GCF_DIM/2; b>=-GCF_DIM/2 ;b--) {
         PRECISION r2 = gcf[GCF_DIM*GCF_DIM*(GCF_GRID*sub_y+sub_x) + 
                        GCF_DIM*b+a].x;
         PRECISION i2 = gcf[GCF_DIM*GCF_DIM*(GCF_GRID*sub_y+sub_x) + 
                        GCF_DIM*b+a].y;
         PRECISION r1 = in_vals[n].x;
         PRECISION i1 = in_vals[n].y;
         if (main_x+a < 0 || main_y+b < 0 || 
             main_x+a >= IMG_SIZE  || main_y+b >= IMG_SIZE) {
         } else {
            out[main_x+a+IMG_SIZE*(main_y+b)].x += r1*r2-i1*i2; 
            out[main_x+a+IMG_SIZE*(main_y+b)].y += r1*i2+r2*i1;
         }
      }
      //std::cout << "val = " << out[n].r << "+ i" << out[n].i << std::endl;
   } 
   gcf -= GCF_DIM*(GCF_DIM+1)/2;
}
template <class T,class Thalf>
int w_comp_main(const void* A, const void* B) {
   Thalf quota, rema, quotb, remb;
   rema = modf((*((T*)A)).x, &quota);
   remb = modf((*((T*)B)).x, &quotb);
   if (quota > quotb) return 1;
   if (quota < quotb) return -1;
   else {
     rema = modf((*((T*)A)).y, &quota);
     remb = modf((*((T*)B)).y, &quotb);
     if (quota > quotb) return 1;
     if (quota < quotb) return -1;
     else return 0;
   }
   return 0;
}
template <class T,class Thalf>
int w_comp_sub(const void* A, const void* B) {
   Thalf quota, rema, quotb, remb;
   rema = modf((*((T*)A)).x, &quota);
   remb = modf((*((T*)B)).x, &quotb);
   int sub_xa = (int) (GCF_GRID*rema);
   int sub_xb = (int) (GCF_GRID*remb);
   rema = modf((*((T*)A)).y, &quota);
   remb = modf((*((T*)B)).y, &quotb);
   int suba = (int) (GCF_GRID*rema) + GCF_GRID*sub_xa;
   int subb = (int) (GCF_GRID*remb) + GCF_GRID*sub_xb;
   if (suba > subb) return 1;
   if (suba < subb) return -1;
   return 0;
}
template <class T,class Thalf>
int w_comp_full(const void* A, const void* B) {
   int result = w_comp_sub<T,Thalf>(A,B);
   if (0==result) return w_comp_main<T,Thalf>(A,B);
   else return result;
}
#if 0
struct comp_grid {
   int blockgrid, blocksize;
   public:
   comp_grid(int img_dim, int gcf_dim) {
      blocksize = gcf_dim/2;
      blockgrid = img_dim/blocksize;
   }
   int __cdecl operator () (const void* A, const void* B) const {
      int gridxa = (*(int2*)A).x/GCF_GRID;
      int gridxb = (*(int2*)B).x/GCF_GRID;
      int gridya = (*(int2*)A).y/GCF_GRID;
      int gridyb = (*(int2*)B).y/GCF_GRID;
      if (gridya > gridyb) return 1;
      if (gridya < gridyb) return -1;
      if (gridxa > gridxb) return 1;
      if (gridxa < gridxb) return  -1;
      int suba = GCF_GRID*((*(int2*)A).x%GCF_GRID) + (*(int2*)A).y%GCF_GRID;
      int subb = GCF_GRID*((*(int2*)B).x%GCF_GRID) + (*(int2*)B).y%GCF_GRID;
      if (suba > subb) return 1;
      if (suba < subb) return -1;
      return  0;
   }
};
#else
template <class T, class Thalf>
int comp_grid (const void* A, const void* B) {
      int blocksize = GCF_DIM/2;
      int mainxa = floorf((*(T*)A).x);
      int mainxb = floorf((*(T*)B).x);
      int mainya = floorf((*(T*)A).y);
      int mainyb = floorf((*(T*)B).y);
      int gridxa = mainxa/blocksize;
      int gridxb = mainxb/blocksize;
      int gridya = mainya/blocksize;
      int gridyb = mainyb/blocksize;
      if (gridya*IMG_SIZE/blocksize+gridxa > 
          gridyb*IMG_SIZE/blocksize+gridxb) return 1;
      if (gridya*IMG_SIZE/blocksize+gridxa < 
          gridyb*IMG_SIZE/blocksize+gridxb) return -1;
      Thalf suba = GCF_GRID*((*(T*)A).x-mainxa) + (*(T*)A).y-mainya;
      Thalf subb = GCF_GRID*((*(T*)B).x-mainxb) + (*(T*)B).y-mainyb;
      if (suba > subb) return 1;
      if (suba < subb) return -1;
      return  0;
}
#endif


int main(void) {

#ifdef __MANAGED
   PRECISION2* out, *in, *in_vals, *gcf;
   std::cout << "mallocManaged out" << std::endl;
   hipMallocManaged(&out, sizeof(PRECISION2)*(IMG_SIZE*IMG_SIZE+2*IMG_SIZE*GCF_DIM+2*GCF_DIM));
   hipMallocManaged(&in, sizeof(PRECISION2)*NPOINTS);
   hipMallocManaged(&in_vals, sizeof(PRECISION2)*NPOINTS);
   hipMallocManaged(&gcf, sizeof(PRECISION2)*64*GCF_DIM*GCF_DIM);
#else
   std::cout << "malloc out" << std::endl;
   PRECISION2* out = (PRECISION2*) malloc(sizeof(PRECISION2)*(IMG_SIZE*IMG_SIZE+2*IMG_SIZE*GCF_DIM+2*GCF_DIM));
   std::cout << "out = " << out << std::endl;
   PRECISION2* in = (PRECISION2*) malloc(sizeof(PRECISION2)*NPOINTS);
   PRECISION2* in_vals = (PRECISION2*) malloc(sizeof(PRECISION2)*NPOINTS);

   PRECISION2 *gcf = (PRECISION2*) malloc(64*GCF_DIM*GCF_DIM*sizeof(PRECISION2));
#endif

   init_gcf(gcf, GCF_DIM);
   srand(2541617);
   for(size_t n=0; n<NPOINTS; n++) {
      in[n].x = ((float)rand())/RAND_MAX*IMG_SIZE;
      in[n].y = ((float)rand())/RAND_MAX*IMG_SIZE;
      in_vals[n].x = ((float)rand())/RAND_MAX;
      in_vals[n].y = ((float)rand())/RAND_MAX;
   }
   //Zero the data in the offset areas
   for (int x=-IMG_SIZE*GCF_DIM-GCF_DIM;x<0;x++) {
      out[x].x = 0.0; out[x].y = 0.0;
   }
   for (int x=0;x<IMG_SIZE*GCF_DIM+GCF_DIM;x++) {
      out[x+IMG_SIZE*IMG_SIZE].x = 0.0; out[x+IMG_SIZE*IMG_SIZE].y = 0.0;
   }

#ifdef __GATHER
   std::qsort(in, NPOINTS, sizeof(PRECISION2), comp_grid<PRECISION2,PRECISION>);
#else
#ifdef __MOVING_WINDOW
   std::qsort(in, NPOINTS, sizeof(PRECISION2), w_comp_main<PRECISION2,PRECISION>);
#else
   std::qsort(in, NPOINTS, sizeof(PRECISION2), w_comp_sub<PRECISION2,PRECISION>);
#endif
#endif
   std::cout << "sorted" << std::endl;
   
   std::cout << "out = " << out << std::endl;
   gridGPU(out,in,in_vals,NPOINTS,IMG_SIZE,gcf,GCF_DIM);
   std::cout << "out = " << out << std::endl;
#ifdef __CPU_CHECK
   std::cout << "Computing on CPU..." << std::endl;
   PRECISION2 *out_cpu=(PRECISION2*)malloc(sizeof(PRECISION2)*IMG_SIZE*IMG_SIZE);
   memset(out_cpu, 0, sizeof(PRECISION2)*IMG_SIZE*IMG_SIZE);
   
   gridCPU(out_cpu+IMG_SIZE*GCF_DIM+GCF_DIM,in,in_vals,NPOINTS,IMG_SIZE,gcf,GCF_DIM);
#endif
   std::cout << "out = " << out << std::endl;


#ifdef __CPU_CHECK
   std::cout << "Checking results against CPU:" << std::endl;
   for (size_t yy = 0; yy < IMG_SIZE; yy++) {
   for (size_t xx = 0; xx < IMG_SIZE; xx++) {
     int n = GCF_DIM+IMG_SIZE*GCF_DIM+yy*IMG_SIZE+xx;
     if (fabs(out[n].x-out_cpu[n].x) > 0.0000001 ||
         fabs(out[n].y-out_cpu[n].y) > 0.0000001 )
        std::cout << xx << ", " << yy << " : " 
                  << out[n].x << ", " << out[n].y 
                  << " vs. " << out_cpu[n].x << ", " << out_cpu[n].y 
                  << std::endl;
   }
   }
   //std::cout << "free out_cpu" << std::endl;
   //free(out_cpu);out_cpu=NULL;
#endif
   std::cout << "out = " << out << std::endl;
#ifdef __MANAGED
   std::cout << "hipFree out" << std::endl;
   hipFree(out);out=NULL;
   hipFree(in);in=NULL;
   hipFree(in_vals);in_vals=NULL;
   hipFree(gcf);gcf=NULL;
#else
   std::cout << "out = " << out << std::endl;
   std::cout << "free out" << std::endl;
   free(out);out=NULL;
   std::cout << "out = " << out << std::endl;
   free(in);in=NULL;
   free(in_vals);in_vals=NULL;
   free(gcf);gcf=NULL;
#endif
}
